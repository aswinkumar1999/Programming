#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
using namespace std;
#define N 8000

__global__
void allzero(int *arr){
  arr[threadIdx.x]=0;
}

__global__
void additoarray(int *arr){
  arr[blockIdx.x*blockDim.x+threadIdx.x]=blockIdx.x*blockDim.x+threadIdx.x;
}
//Removing Thread Divergence
//Pseudo code
//Q : // assert(x == y || x == z);if (x == y) x = z;else x = y;
__global__
void no_thread_divergence(){
  x= (x==y||x==z)?z:y;
}

int main(void) {
  int *arr;
  hipMallocManaged(&arr,N*sizeof(int));
  int threadsperblock=256;
  int numblocks = (N/threadsperblock)+1;
  allzero <<<numblocks,threadsperblock>>>(arr);
  hipDeviceSynchronize();
  additoarray<<<numblocks,threadsperblock>>>(arr);
  hipDeviceSynchronize();
  for(int i=0;i<N;i++){
    cout<<arr[i]<<"\t";
    
  }
}
