
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to generate random numbers
__global__
void genran(int *rnd,double m)
{
  double n,a=1103515245, c=12345;
   n=blockIdx.x*blockDim.x+threadIdx.x;
   //n=threadIdx.x;
   for(int i=0;i<threadIdx.x;i++)
   n=fmod(((n*a)+c),m);
   __syncthreads();
   atomicAdd(&rnd[(unsigned long int)n],1);
}


int main(void)
{
  int t=29;
  long int m = pow(2,t);
  int *rnd;
  double val;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&rnd, m*sizeof(int));

  // initialize
    val = m;
  for (int i = 0; i < m; i++) {
    rnd[i] = 0;
  }

  //generate random numbers
  int blockSize = 128;
  int numblocks = (m+blockSize-1)/blockSize;
  // Run kernel
  genran<<<numblocks, blockSize>>>(rnd,val);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  //Generate Histrogram
  long double count =0,j=0;
  for(long int i=0;i<m;++i)
  {
   count+=rnd[i];
   j++;
   if(j==pow(2,t-5))
   {
     j=0;
     printf("|");
     count/=pow(2,t-10);
     for(int k=0;k<count;++k)
     printf("*");
     printf("\n");
     count=0;
   }
  }
  // Free memory
  hipFree(rnd);
  return 0;
}
